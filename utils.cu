#include "hip/hip_runtime.h"
#include "utils.cuh"



//struct point_transform_for_rotation
//{
//    const Eigen::Matrix3f _rotation;
//
//    explicit point_transform_for_rotation(Eigen::Matrix3f rotation):_rotation(rotation){}
//
//    __host__ __device__
//
//    Eigen::Vector3f operator()(const Eigen::Vector3f& dev_point)
//    {
//        return (dev_point[0] * _rotation.col(0) / 0.02 + dev_point[1] * _rotation.col(1) / 0.02 + dev_point[2] * _rotation.col(2) / 0.02);
//    }
//};
//
//
//thrust::device_vector<Eigen::Vector3f> CloudTransform(thrust::device_vector<Eigen::Vector3f> cloud, const Eigen::Matrix3f& rotation)
//{
//    thrust::device_vector<Eigen::Vector3f> transformed_cloud(cloud.size());
//    thrust::transform(thrust::device, cloud.begin(), cloud.end(), transformed_cloud.begin(), point_transform_for_rotation(rotation));
//    return transformed_cloud;
//}

struct get_rotation
{
    __host__ __device__
    Eigen::Matrix3f operator()(const Rigid3f& pose)
    {
        Eigen::Vector3f rpy = pose._rotation;

        float alpha = rpy[0];
        float beta  = rpy[1];
        float gamma = rpy[2];

        Eigen::Matrix3f rotation;
        rotation << cosf(alpha)*cosf(gamma)-cosf(beta)*sinf(alpha)*sinf(gamma), -cosf(beta)*cosf(gamma)*sinf(alpha)-cosf(alpha)*sinf(gamma),  sinf(alpha)*sinf(beta),
                    cosf(gamma)*sinf(alpha)+cosf(alpha)*cosf(beta)*sinf(gamma),  cosf(alpha)*cosf(beta)*cosf(gamma)-sinf(alpha)*sinf(gamma), -cosf(alpha)*sinf(beta),
                    sinf(beta)*sinf(gamma),                                      cosf(gamma)*sinf(beta),                                      cosf(beta);
        return rotation;
    }
};


struct get_transform
{
    __host__ __device__
    Eigen::Matrix4f operator()(const Rigid3f& pose)
    {
        Eigen::Vector3f rpy = pose._rotation;
        Eigen::Vector3f xyz = pose._translation;

        float alpha = rpy[0];
        float beta  = rpy[1];
        float gamma = rpy[2];

        float x = xyz[0];
        float y = xyz[1];
        float z = xyz[2];

        Eigen::Matrix4f transform;
        transform << cosf(alpha)*cosf(gamma)-cosf(beta)*sinf(alpha)*sinf(gamma), -cosf(beta)*cosf(gamma)*sinf(alpha)-cosf(alpha)*sinf(gamma),  sinf(alpha)*sinf(beta),   x,
                     cosf(gamma)*sinf(alpha)+cosf(alpha)*cosf(beta)*sinf(gamma),  cosf(alpha)*cosf(beta)*cosf(gamma)-sinf(alpha)*sinf(gamma), -cosf(alpha)*sinf(beta),   y,
                     sinf(beta)*sinf(gamma),                                      cosf(gamma)*sinf(beta),                                      cosf(beta),               z,
                     0.0,                                                         0.0,                                                         0.0,                    1.0;
        return transform;
    }
};


struct point_transform
{
    const Eigen::Vector3f _point;

    explicit point_transform(Eigen::Vector3f point):_point(point){}

    __host__ __device__
    Eigen::Vector3f operator()(const Eigen::Matrix3f& rotation)
    {
        return (_point[0] * rotation.col(0) + _point[1] * rotation.col(1) + _point[2] * rotation.col(2));
//        return (_point[0] * rotation.col(0) / 0.02 + _point[1] * rotation.col(1) / 0.02 + _point[2] * rotation.col(2) / 0.02);
    }
};

struct eigen_compare
{
    __host__ __device__
    bool operator()(const Eigen::Vector3f& lhs, const Eigen::Vector3f& rhs)
    {
        return (lhs[0]<rhs[0])||(fabs(lhs[0]-rhs[0])<1e-6&&lhs[1]<rhs[1])||((lhs[0]-rhs[0])<1e-6&&fabs(lhs[1]-rhs[1])<1e-6&&lhs[2]<rhs[2]);
    }
};


__host__ __device__ bool operator==(const Eigen::Vector3f& lhs, const Eigen::Vector3f& rhs)
{
    return fabs(lhs[0]-rhs[0])<1e-6&&fabs(lhs[1]-rhs[1])<1e-6&&fabs(lhs[2]-rhs[2])<1e-6;
}

__host__ __device__ bool operator>(const Eigen::Vector3f& lhs, const Eigen::Vector3f& rhs)
{
    return (lhs[0]>rhs[0])||(fabs(lhs[0]-rhs[0])<1e-6&&lhs[1]>rhs[1])||((lhs[0]-rhs[0])<1e-6&&fabs(lhs[1]-rhs[1])<1e-6&&lhs[2]>rhs[2]);
}

__host__ __device__ int BinarySearchRecursive(const Eigen::Vector3f* points, int low, int high, Eigen::Vector3f point)
{
    if (low > high)
        return -1;
    int mid = low + (high - low) / 2;
    if (points[mid] == point)
        return mid;
    else if (points[mid] > point)
        return BinarySearchRecursive(points, low, mid - 1, point);
    else
        return BinarySearchRecursive(points, mid + 1, high, point);
}


struct match
{
    const Eigen::Vector3f* _map;
    const int _size;

    explicit match(Eigen::Vector3f* map, int size):_map(map), _size(size){}

    __host__ __device__
    float operator()(const Eigen::Vector3f& point)
    {
        int idx = BinarySearchRecursive(_map, 0, _size-1, point);
//        printf("%f, %f, %f: %d\n", point[0], point[1], point[2], idx);
        if(idx < 0)
        {
            return 0.0;
        }
        else
        {
//            return _map[idx][3];
            return 1.0;
        }
    }
};


struct compute_score
{
    const int _size;
    explicit compute_score(int size):_size(size){}

    __host__ __device__
    float operator()(const float& sum)
    {
        return float(sum/float(_size));
    }
};







//struct cloud_transform
//{
//    const thrust::device_vector<Eigen::Vector3f> _cloud;
//
//    explicit cloud_transform(thrust::device_vector<Eigen::Vector3f> cloud):_cloud(cloud){}
//
//    __host__ __device__ thrust::device_vector<Eigen::Vector3f> operator()(const Eigen::Matrix3f& dev_rotation)
//    {
//        thrust::device_vector<Eigen::Vector3f> transformed_cloud(_cloud.size());
//        thrust::transform(thrust::device, _cloud.begin(), _cloud.end(), transformed_cloud.begin(), point_transform(dev_rotation));
//        return transformed_cloud;
//    }
//};
//
//
//thrust::device_vector< thrust::device_vector<Eigen::Vector3f> > MultiCloudsTransform(thrust::device_vector<Eigen::Matrix3f> rotations, const thrust::device_vector<Eigen::Vector3f>& cloud)
//{
//    thrust::device_vector< thrust::device_vector<Eigen::Vector3f> > transformed_clouds(rotations.size());
//    thrust::transform(thrust::device, rotations.begin(), rotations.end(), transformed_clouds.begin(), cloud_transform(cloud));
//    return transformed_clouds;
//}



//struct exp_functor
//{
//    template<typename T>
//    __host__ __device__
//    thrust::complex<T> operator()(const thrust::complex<T> &x)
//    {
//        return exp(x);
//    } // end operator()()
//}; // end make_pair_functor




//__global__ void Kernel(Eigen::Vector3f* dev_submap, Eigen::Vector3f* dev_target, float* dev_scores)
//{
//    int i = blockIdx.x;
//    dev_scores[i] = dev_submap->col(0)[0];
//}





int GetOptPoseIndex(const std::vector<Eigen::Vector3f>& submap, const std::vector<Eigen::Vector3f>& map, const std::vector<Rigid3f>& poses)
{
//    thrust::device_vector<Eigen::Vector3f> dev_submap = submap;
//
//    float scores[100];
//    float* dev_scores;
//
//    hipMalloc(&dev_scores,100* sizeof(float));
//
//    Kernel<<<100,1>>>(thrust::raw_pointer_cast(&dev_submap[0]), thrust::raw_pointer_cast(&dev_submap[0]), dev_scores);
//
//    hipMemcpy(scores, dev_scores, 100* sizeof(float), hipMemcpyDeviceToHost);
//
//    hipFree(dev_scores);
//
//    hipDeviceSynchronize();
//
//    for(int i = 0; i < 100; i++)
//    {
//        std::cout<<scores[i]<<std::endl;
//    }


//    int poses_num = 100000;

//    Eigen::Matrix3f rotation = Eigen::Matrix3f::Identity();
//    thrust::device_vector<Eigen::Vector3f> dev_submap = submap;
//    thrust::device_vector<Eigen::Vector3f> dev_trans_submap = CloudTransform(dev_submap, rotation);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    thrust::device_vector<Rigid3f> dev_poses = poses;
    thrust::device_vector<Eigen::Matrix3f> dev_rotations(poses.size());
//    thrust::fill(thrust::device, dev_rotations.begin(), dev_rotations.end(), rotation);
    thrust::transform(thrust::device, dev_poses.begin(), dev_poses.end(), dev_rotations.begin(), get_rotation());
    hipDeviceSynchronize();

    std::cout<<"rotations acquired"<<std::endl;
    std::cout<<"pose num:"<<dev_rotations.size()<<std::endl;



    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time to generate:  %3.1f ms \n", time);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);



    thrust::device_vector<Eigen::Vector3f> trans_point(poses.size());

    thrust::device_vector<float> score_tile(poses.size());
    thrust::device_vector<float> score_bins(poses.size());
    thrust::fill(thrust::device, score_bins.begin(), score_bins.end(), 0.0);
    hipDeviceSynchronize();

    int map_size = map.size();
    int submap_size = submap.size();
    thrust::device_vector<Eigen::Vector3f> dev_map = map;

    thrust::sort(thrust::device, dev_map.begin(), dev_map.end(), eigen_compare());
    hipDeviceSynchronize();

//    thrust::host_vector<Eigen::Vector3f> sorted_map = dev_map;
//    for(int i = 0; i < sorted_map.size(); i++)
//    {
//        std::cout<<sorted_map[i]<<std::endl;
//        std::cout<<std::endl;
//    }

//    thrust::device_ptr<Eigen::Vector3f> dev_map_ptr = &dev_map[0];



//    clock_t start = clock();


    for(int i = 0 ; i < submap.size(); i++)
    {
        thrust::transform(thrust::device, dev_rotations.begin(), dev_rotations.end(), trans_point.begin(), point_transform(submap[i]));
        hipDeviceSynchronize();
        thrust::transform(thrust::device, trans_point.begin(), trans_point.end(), score_tile.begin(), match(thrust::raw_pointer_cast(&dev_map[0]), map_size));
        hipDeviceSynchronize();
        thrust::transform(thrust::device, score_bins.begin(), score_bins.end(), score_tile.begin(), score_bins.begin(), thrust::plus<float>());
        hipDeviceSynchronize();
    }
    thrust::transform(thrust::device, score_bins.begin(), score_bins.end(), score_bins.begin(), compute_score(submap_size));
    hipDeviceSynchronize();

    thrust::device_vector<float>::iterator max_element_iter = thrust::max_element(score_bins.begin(), score_bins.end());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time to generate:  %3.1f ms \n", time);

//    clock_t during = clock() - start;
//    std::cout<<"GPU: "<<double(during / 1000.0)<<std::endl;

    int opt_pose_idx = max_element_iter - score_bins.begin();

//    std::cout<<"opt pose index: "<<opt_pose_idx<<std::endl;
//    std::cout<<"opt pose score: "<<score_bins[opt_pose_idx]<<std::endl;
//    std::cout<<"opt pose: "<<poses[opt_pose_idx]._rotation<<std::endl;

    int count = 0;
    for(int i = 0; i < score_bins.size(); i++)
    {
        if(score_bins[i]==1.0)
        {
            std::cout<<"pose: "<<poses[i]._rotation<<std::endl;
        }
    }

    return opt_pose_idx;
}