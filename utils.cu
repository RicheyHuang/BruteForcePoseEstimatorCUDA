#include "hip/hip_runtime.h"
#include "utils.cuh"


struct get_rotation
{
    __host__ __device__
    Eigen::Matrix3f operator()(const Rigid3f& pose)
    {
        Eigen::Vector3f rpy = pose._rotation;

        float alpha = rpy[0];
        float beta  = rpy[1];
        float gamma = rpy[2];

        Eigen::Matrix3f rotation;

//      ZYX order
        rotation << cosf(beta)*cosf(gamma),                                     -cosf(beta)*sinf(gamma),                                        sinf(beta),
                    sinf(alpha)*sinf(beta)*cosf(gamma)+cosf(alpha)*sinf(gamma), -sinf(alpha)*sinf(beta)*sinf(gamma)+cosf(alpha)*cosf(gamma),   -sinf(alpha)*cosf(beta),
                   -cosf(alpha)*sinf(beta)*cosf(gamma)+sinf(alpha)*sinf(gamma),  cosf(alpha)*sinf(beta)*sinf(gamma)+sinf(alpha)*cosf(gamma),    cosf(alpha)*cosf(beta);

//      XYZ order
//        rotation << cosf(beta)*cosf(alpha),     sinf(gamma)*sinf(beta)*cosf(alpha)-cosf(gamma)*sinf(alpha),    cosf(gamma)*sinf(beta)*cosf(alpha)+sinf(gamma)*sinf(alpha),
//                    cosf(beta)*sinf(alpha),     sinf(gamma)*sinf(beta)*sinf(alpha)+cosf(gamma)*cosf(alpha),    cosf(gamma)*sinf(beta)*sinf(alpha)-sinf(gamma)*cosf(alpha),
//                   -sinf(beta),                 sinf(gamma)*cosf(beta),                                        cosf(gamma)*cosf(beta);

        return rotation;
    }
};


struct get_transform
{
    __host__ __device__
    Eigen::Matrix4f operator()(const Rigid3f& pose)
    {
        Eigen::Vector3f rpy = pose._rotation;
        Eigen::Vector3f xyz = pose._translation;

        float alpha = rpy[0];
        float beta  = rpy[1];
        float gamma = rpy[2];

        float x = xyz[0];
        float y = xyz[1];
        float z = xyz[2];

        Eigen::Matrix4f transform;

//      ZYX order
        transform << cosf(beta)*cosf(gamma),                                     -cosf(beta)*sinf(gamma),                                        sinf(beta),              x,
                     sinf(alpha)*sinf(beta)*cosf(gamma)+cosf(alpha)*sinf(gamma), -sinf(alpha)*sinf(beta)*sinf(gamma)+cosf(alpha)*cosf(gamma),   -sinf(alpha)*cosf(beta),  y,
                    -cosf(alpha)*sinf(beta)*cosf(gamma)+sinf(alpha)*sinf(gamma),  cosf(alpha)*sinf(beta)*sinf(gamma)+sinf(alpha)*cosf(gamma),    cosf(alpha)*cosf(beta),  z,
                     0.0,                                                         0.0,                                                           0.0,                     1.0;

//      XYZ order
//        transform << cosf(beta)*cosf(alpha),     sinf(gamma)*sinf(beta)*cosf(alpha)-cosf(gamma)*sinf(alpha),    cosf(gamma)*sinf(beta)*cosf(alpha)+sinf(gamma)*sinf(alpha),  x,
//                     cosf(beta)*sinf(alpha),     sinf(gamma)*sinf(beta)*sinf(alpha)+cosf(gamma)*cosf(alpha),    cosf(gamma)*sinf(beta)*sinf(alpha)-sinf(gamma)*cosf(alpha),  y,
//                    -sinf(beta),                 sinf(gamma)*cosf(beta),                                        cosf(gamma)*cosf(beta),                                      z,
//                     0.0,                        0.0,                                                           0.0,                                                         1.0;

return transform;
    }
};


struct point_transform
{
    const Eigen::Vector3f _point;

    explicit point_transform(Eigen::Vector3f point):_point(point){}

    __host__ __device__
    Eigen::Vector3f operator()(const Eigen::Matrix3f& rotation)
    {
        return (_point[0] * rotation.col(0) + _point[1] * rotation.col(1) + _point[2] * rotation.col(2));
//        return (_point[0] * rotation.col(0) / 0.02 + _point[1] * rotation.col(1) / 0.02 + _point[2] * rotation.col(2) / 0.02);
    }
};

struct eigen_compare
{
    __host__ __device__
    bool operator()(const Eigen::Vector3f& lhs, const Eigen::Vector3f& rhs)
    {
        return (lhs[0]<rhs[0])||(fabs(lhs[0]-rhs[0])<1e-6&&lhs[1]<rhs[1])||((lhs[0]-rhs[0])<1e-6&&fabs(lhs[1]-rhs[1])<1e-6&&lhs[2]<rhs[2]);
    }
};


__host__ __device__ bool operator==(const Eigen::Vector3f& lhs, const Eigen::Vector3f& rhs)
{
    return fabs(lhs[0]-rhs[0])<1e-6&&fabs(lhs[1]-rhs[1])<1e-6&&fabs(lhs[2]-rhs[2])<1e-6;
}

__host__ __device__ bool operator>(const Eigen::Vector3f& lhs, const Eigen::Vector3f& rhs)
{
    return (lhs[0]>rhs[0])||(fabs(lhs[0]-rhs[0])<1e-6&&lhs[1]>rhs[1])||((lhs[0]-rhs[0])<1e-6&&fabs(lhs[1]-rhs[1])<1e-6&&lhs[2]>rhs[2]);
}

__host__ __device__ int BinarySearchRecursive(const Eigen::Vector3f* points, int low, int high, Eigen::Vector3f point)
{
    if (low > high)
        return -1;
    int mid = low + (high - low) / 2;
    if (points[mid] == point)
        return mid;
    else if (points[mid] > point)
        return BinarySearchRecursive(points, low, mid - 1, point);
    else
        return BinarySearchRecursive(points, mid + 1, high, point);
}


struct match
{
    const Eigen::Vector3f* _map;
    const int _size;

    explicit match(Eigen::Vector3f* map, int size):_map(map), _size(size){}

    __host__ __device__
    float operator()(const Eigen::Vector3f& point)
    {
        int idx = BinarySearchRecursive(_map, 0, _size-1, point);
//        printf("%f, %f, %f: %d\n", point[0], point[1], point[2], idx);
        if(idx < 0)
        {
            return 0.0;
        }
        else
        {
//            return _map[idx][3];
            return 1.0;
        }
    }
};


struct compute_score
{
    const int _size;
    explicit compute_score(int size):_size(size){}

    __host__ __device__
    float operator()(const float& sum)
    {
        return float(sum/float(_size));
    }
};


struct get_unit_pose
{
    const int _offset;
    const float _resolution;
    explicit get_unit_pose(const int& offset, const float& resolution):_offset(offset), _resolution(resolution){}

    __host__ __device__
    float operator()(int index)
    {
        return float((index-_offset)*_resolution);
    }
};

struct get_6dof
{
    const int _loop_size_pyxyz;
    const int _loop_size_yxyz;
    const int _loop_size_xyz;
    const int _loop_size_yz;
    const int _loop_size_z;


    const float* _angles;
    const float* _displacements;


    explicit get_6dof(const int& loop_size_pyxyz,
                      const int& loop_size_yxyz,
                      const int& loop_size_xyz,
                      const int& loop_size_yz,
                      const int& loop_size_z,
                      const float* displacements,
                      const float* angles):
                      _loop_size_pyxyz(loop_size_pyxyz),
                      _loop_size_yxyz(loop_size_yxyz),
                      _loop_size_xyz(loop_size_xyz),
                      _loop_size_yz(loop_size_yz),
                      _loop_size_z(loop_size_z),
                      _angles(angles),
                      _displacements(displacements){}

    __host__ __device__
    Eigen::Matrix<float, 6, 1> operator()(int pose_index)
    {
        Eigen::Matrix<float, 6, 1> pose;

        pose(0, 0) = _angles[int(pose_index/_loop_size_pyxyz)];
        pose(1, 0) = _angles[int(pose_index%_loop_size_pyxyz/_loop_size_yxyz)];
        pose(2, 0) = _angles[int(pose_index%_loop_size_pyxyz%_loop_size_yxyz/_loop_size_xyz)];
        pose(3, 0) = _displacements[int(pose_index%_loop_size_pyxyz%_loop_size_yxyz%_loop_size_xyz/_loop_size_yz)];
        pose(4, 0) = _displacements[int(pose_index%_loop_size_pyxyz%_loop_size_yxyz%_loop_size_xyz%_loop_size_yz/_loop_size_z)];
        pose(5, 0) = _displacements[int(pose_index%_loop_size_pyxyz%_loop_size_yxyz%_loop_size_xyz%_loop_size_yz%_loop_size_z)];

        return pose;
    }
};


int GetOptPoseIndex(const std::vector<Eigen::Vector3f>& submap, const std::vector<Eigen::Vector3f>& map, const std::vector<Rigid3f>& poses)
{
//    float time;
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    hipEventRecord(start, 0);
//
//    int linear_winsize = 2;
//    float linear_step = 0.02;
//    int linear_space_size = 2*linear_winsize+1;
//    thrust::device_vector<int> linear_indices(linear_space_size);
//    thrust::sequence(linear_indices.begin(), linear_indices.end());
//    hipDeviceSynchronize();
//    thrust::device_vector<float> displacements(linear_space_size);
//    thrust::transform(linear_indices.begin(), linear_indices.end(), displacements.begin(), get_unit_pose(linear_winsize, linear_step));
//    hipDeviceSynchronize();
//
//    int angular_winsize = 2;
//    float angular_step = 0.01;
//    int angular_space_size = 2*angular_winsize+1;
//    thrust::device_vector<int> angular_indices(angular_space_size);
//    thrust::sequence(angular_indices.begin(), angular_indices.end());
//    hipDeviceSynchronize();
//    thrust::device_vector<float> angles(angular_space_size);
//    thrust::transform(angular_indices.begin(), angular_indices.end(), angles.begin(), get_unit_pose(angular_winsize, angular_step));
//    hipDeviceSynchronize();
//
//    int pose_num = int(pow(angular_space_size,3)*pow(linear_space_size, 3));
//    thrust::device_vector<Eigen::Matrix<float, 6, 1> > poses(pose_num);
//
//    thrust::device_vector<int> pose_indices(pose_num);
//    thrust::sequence(pose_indices.begin(), pose_indices.end());
//    hipDeviceSynchronize();
//
//    int loop_size_pyxyz = int(pow(angular_space_size,2)*pow(linear_space_size, 3));
//    int loop_size_yxyz = int(angular_space_size*pow(linear_space_size, 3));
//    int loop_size_xyz = int(pow(linear_space_size, 3));
//    int loop_size_yz = int(pow(linear_space_size, 2));
//    int loop_size_z = int(linear_space_size);
//
//    thrust::transform(thrust::device, pose_indices.begin(), pose_indices.end(), poses.begin(), get_6dof(loop_size_pyxyz, loop_size_yxyz, loop_size_xyz,
//                      loop_size_yz, loop_size_z, thrust::raw_pointer_cast(&angles[0]), thrust::raw_pointer_cast(&displacements[0])));
//    hipDeviceSynchronize();
//
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&time, start, stop);
//    printf("Time to generate:  %3.1f ms \n", time);
//
//
////    thrust::host_vector<Eigen::Matrix<float, 6, 1> > host = poses;
////    for(int i = 0; i < host.size(); i++)
////    {
////        std::cout<<host[i][0]<<", "<<host[i][1]<<", "<<host[i][2]<<", "<<host[i][3]<<", "<<host[i][4]<<", "<<host[i][5]<<", "<<std::endl;
////    }
//
//    return 0;






    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    thrust::device_vector<Rigid3f> dev_poses = poses;
    thrust::device_vector<Eigen::Matrix3f> dev_rotations(poses.size());
//    thrust::fill(thrust::device, dev_rotations.begin(), dev_rotations.end(), rotation);
    thrust::transform(thrust::device, dev_poses.begin(), dev_poses.end(), dev_rotations.begin(), get_rotation());
    hipDeviceSynchronize();

    std::cout<<"rotations acquired"<<std::endl;
    std::cout<<"pose num:"<<dev_rotations.size()<<std::endl;



    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time to generate:  %3.1f ms \n", time);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);



    thrust::device_vector<Eigen::Vector3f> trans_point(poses.size());

    thrust::device_vector<float> score_tile(poses.size());
    thrust::device_vector<float> score_bins(poses.size());
    thrust::fill(thrust::device, score_bins.begin(), score_bins.end(), 0.0);
    hipDeviceSynchronize();

    int map_size = map.size();
    int submap_size = submap.size();
    thrust::device_vector<Eigen::Vector3f> dev_map = map;

    thrust::sort(thrust::device, dev_map.begin(), dev_map.end(), eigen_compare());
    hipDeviceSynchronize();



    for(int i = 0 ; i < submap.size(); i++)
    {
        thrust::transform(thrust::device, dev_rotations.begin(), dev_rotations.end(), trans_point.begin(), point_transform(submap[i]));
        hipDeviceSynchronize();
        thrust::transform(thrust::device, trans_point.begin(), trans_point.end(), score_tile.begin(), match(thrust::raw_pointer_cast(&dev_map[0]), map_size));
        hipDeviceSynchronize();
        thrust::transform(thrust::device, score_bins.begin(), score_bins.end(), score_tile.begin(), score_bins.begin(), thrust::plus<float>());
        hipDeviceSynchronize();
    }
    thrust::transform(thrust::device, score_bins.begin(), score_bins.end(), score_bins.begin(), compute_score(submap_size));
    hipDeviceSynchronize();

    thrust::device_vector<float>::iterator max_element_iter = thrust::max_element(score_bins.begin(), score_bins.end());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time to generate:  %3.1f ms \n", time);


    int opt_pose_idx = max_element_iter - score_bins.begin();

    std::cout<<"opt pose index: "<<opt_pose_idx<<std::endl;
    std::cout<<"opt pose score: "<<score_bins[opt_pose_idx]<<std::endl;
    std::cout<<"opt pose: "<<poses[opt_pose_idx]._rotation<<std::endl;

    return opt_pose_idx;
}